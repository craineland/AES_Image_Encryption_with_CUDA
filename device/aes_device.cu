#include "hip/hip_runtime.h"
// System Imports
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Project Imports
#include "tools/bitmap.c"
#include "tools/helper_functions.cu"
#include "tools/round_keys.cu"
#include "tools/sbox.cu"

/*
 * Parallel implementation of Advanced Encryption Standard (AES)
 * Based on "aes.c" (serial version) by Dani Huertas
 *
 * Encryption/Decryption process simplified with integrated
 * byte substitution and additive key rounds
 *
 * @author      Camden Landis (craine)
 * @version     06.12.2021
 *              DD.MM.YYYY
 */
// =========================================================================

#define BLOCK_DIM  16
#define THREAD_NUM 512

// Number of columns (32-bit words) comprising the State.
// For this standard, num. of col. = 4.
__device__ static int col_num = 4;

/*
 * Transformation in the Cipher that processes the State by cyclically 
 * shifting the last three rows of the State by different offsets. 
 */
__device__ void shift_rows(unsigned char *state) {
    unsigned char i, j, k, state_update;
    for (i = 1; i < 4; i++) {
        k = 0;
        while (k < i) {
            state_update = state[col_num * i + 0];

            for (j = 1; j < col_num; j++) {
                state[col_num * i + j - 1] = state[col_num * i + j];
            }

            state[col_num * i + col_num - 1] = state_update;
            k++;
        }
    }
}

/*
 * Transformation in the Inverse Cipher that is the  
 * cryptographic inverse of shift_rows().
 */
__device__ void inverse_shift_rows(unsigned char *state) {
    unsigned char i, j, k, state_update;
    for (i = 1; i < 4; i++) {
        k = 0;
        while (k < i) {
            state_update = state[col_num * i + col_num - 1];

            for (j = col_num - 1; j > 0; j--) {
                state[col_num * i + j] = state[col_num * i + j - 1];
            }

            state[col_num * i + 0] = state_update;
            k++;
        }
    }
}

/*
 * Transformation in the Cipher that takes all of the columns of the 
 * State and mixes their data (independently of one another) to 
 * produce new columns.
 */
__device__ void mix_columns(unsigned char *state) {
               // a(x) =  {02} + {01}x + {01}x2 + {03}x3
    unsigned char a[] = {0x02,  0x01,   0x01,    0x03};
    unsigned char i, j, col[4], result[4];

    for (j = 0; j < col_num; j++) {
        for (i = 0; i < 4; i++) {
            col[i] = state[col_num * i + j];
        }

        coef_mult(a, col, result);

        for (i = 0; i < 4; i++) {
            state[col_num * i + j] = result[i];
        }
    }
}

/*
 * Transformation in the Inverse Cipher that is the 
 * cryptographic inverse of mix_columns().
 */
__device__ void inverse_mix_columns(unsigned char *state) {
               // a(x) =  {0e} + {09}x + {0d}x2 + {0b}x3
    unsigned char a[] = {0x0e,  0x09,   0x0d,    0x0b};
    unsigned char i, j, col[4], result[4];

    for (j = 0; j < col_num; j++) {
        for (i = 0; i < 4; i++) {
            col[i] = state[col_num * i + j];
        }

        coef_mult(a, col, result);

        for (i = 0; i < 4; i++) {
            state[col_num * i + j] = result[i];
        }
    }
}

/*
 * Performs the simplified AES encryption operation
 */
__global__ void aes_encrypt_naive(unsigned char *image, int size, int key) {
    int t = threadIdx.x;
    int b = blockIdx.x;
    int B = blockDim.x;
    int n = t + b*B;

    if (n < size) {
        // byte substitution
        // index translation from SBOX
        image[n] = sbox[image[n]];
        // shift rows
        shift_rows(image + n);
        // mix columns
        mix_columns(image + n);
        // Add round key
        key_xor(image + n);
    }

    __syncthreads();
}

/*
 * Performs the simplified AES decryption operation
 */
__global__ void aes_decrypt_naive(unsigned char *image, int size, int key) {
    int t = threadIdx.x;
    int b = blockIdx.x;
    int B = blockDim.x;
    int n = t + b*B;

    if (n < size) {
        // inverse add round key
        key_xor(image + n);
        // inverse mix columns
        inverse_mix_columns(image + n);
        // inverse shift rows
        inverse_shift_rows(image + n);
        // inverse byte substitution
        // index translation from Inverse SBOX
        image[n] = inverse_sbox[image[n]];
    }

    __syncthreads();
}


/*
 * Performs the simplified AES encryption operation with shared memory
 */
__global__ void aes_encrypt_shared(unsigned char *image, int size, int key) {
    int t = threadIdx.x;
    int b = blockIdx.x;
    // int B = blockDim.x;
    // int n = t + b*B;

    // shared memory array for state data
    __shared__ unsigned char s_state[THREAD_NUM * BLOCK_DIM];
    int i = 0;

    // shared memory code based on work by 
    // Mengxiao Lin, Jiemin Wu, Xiaorui Wang, Chuyuan Qu 
    // Computer Science @ UC Davis
    // copying image array data to shared memory
    for (int k = t * BLOCK_DIM; k < (t + 1) * BLOCK_DIM; k++) {
        int n_index = k + b * THREAD_NUM * BLOCK_DIM;
        if (n_index < size) {
            s_state[k] = image[n_index];
        }
    }
    __syncthreads();

    // byte substitution
    for (i = t * BLOCK_DIM; i < (t + 1) * BLOCK_DIM; i++) {
        s_state[i] = sbox[s_state[i]];
    }
    __syncthreads();

    // shift rows
    shift_rows(&s_state[t * BLOCK_DIM]);
    __syncthreads();

    // mix columns
    mix_columns(&s_state[t * BLOCK_DIM]);
    __syncthreads();

    // add round key
    key_xor(&s_state[t * BLOCK_DIM]);
    __syncthreads();

    // copying data back to original image array
    for (int k = t * BLOCK_DIM; k < (t + 1) * BLOCK_DIM; k++) {
        int n_index = k + b * THREAD_NUM * BLOCK_DIM;
        if (n_index < size) {
            image[n_index] = s_state[k];
        }
    }
    __syncthreads();
}


/*
 * Performs the simplified AES decryption operation with shared memory
 */
__global__ void aes_decrypt_shared(unsigned char *image, int size, int key) {
    int t = threadIdx.x;
    int b = blockIdx.x;
    int B = blockDim.x;
    int n = t + b*B;

    // shared memory array for state data
    __shared__ unsigned char s_state[THREAD_NUM * BLOCK_DIM];
    int i = 0;

    // shared memory code based on work by 
    // Mengxiao Lin, Jiemin Wu, Xiaorui Wang, Chuyuan Qu 
    // Computer Science @ UC Davis
    // copying encrypted image array data to shared memory
    for (int k = t * BLOCK_DIM; k < (t + 1) * BLOCK_DIM; k++) {
        int n_index = k + b * THREAD_NUM * BLOCK_DIM;
        if (n_index < size)
            s_state[k] = image[n_index];
    }
    __syncthreads();

    // add round key
    key_xor(&s_state[t * BLOCK_DIM]);
    __syncthreads();

    // inverse mix columns
    inverse_mix_columns(&s_state[t * BLOCK_DIM]);
    __syncthreads();

    // inverse byte substitution
    for (i = t * BLOCK_DIM; i < (t + 1) * BLOCK_DIM; i++) {
        s_state[i] = inverse_sbox[s_state[i]];
    }
    __syncthreads();

    // inverse shift rows
    if (n * BLOCK_DIM < size)
        inverse_shift_rows(&s_state[t * BLOCK_DIM]);
    __syncthreads();

    // copying data back to original image array
    for (int k = t * BLOCK_DIM; k < (t + 1) * BLOCK_DIM; k++) {
        int n_index = k + b * THREAD_NUM * BLOCK_DIM;
        if (n_index < size)
            image[n_index] = s_state[k];
    }
    __syncthreads();
}


/*
 * Main function
 *
 * Arguments: 
 *      ./aes_cuda [path to image] [integer key] [path to encrypted output] [path to decrypted output]
 *
 */
int main(int argc, char *argv[]) {
    hipSetDevice(4);

    // argument variables
    char *image_path = argv[1];
    int key = atoi(argv[2]);
    char *encrypted_path = argv[3];
    char *decrypted_path = argv[4];

    if (argc < 5 || argc > 5) {
        printf("Error... Needed Arguments: ./aes_serial [path to image] [integer key] [path to encrypted output] [path to decrypted output]\n");
        return -1;
    }

    printf("\n");
    printf("=== DEVICE Encryption/Decryption Results ===\n");
    printf("\n");

    BITMAPINFOHEADER bitmap_info;
    BITMAPFILEHEADER bitmap_file;

    // load original HOST image
    unsigned char *h_image;
    h_image = LoadBitmapFile(image_path, &bitmap_info, &bitmap_file);
    printf("Size of Input Image: %d%s\n", bitmap_info.biSizeImage, " bytes");
    printf("Dimensions of Image in Pixels (x,y): (%d,%d)\n", bitmap_info.biWidth, bitmap_info.biHeight);

    float encode_elapsed_time, decode_elapsed_time;

    hipEvent_t tic, toc;
    hipEventCreate(&tic);
    hipEventCreate(&toc);

    // store image data in device
    unsigned char *c_image;
    // allocate image in device memory
    hipMalloc((void **)&c_image, bitmap_info.biSizeImage);

    //Copy data from host to device
    hipMemcpy(c_image, h_image, bitmap_info.biSizeImage, hipMemcpyHostToDevice);

    // standard image processing thread-block structure
    // number of thread-blocks B
    int B = ceil(bitmap_info.biSizeImage/(THREAD_NUM*BLOCK_DIM));
    // number of threads T
    int T = THREAD_NUM;

    // dim3 G(ceil((float)bitmap_info.biWidth / (BLOCK_SIZE)), ceil((float)bitmap_info.biHeight / (BLOCK_SIZE)));
    // dim3 B(BLOCK_SIZE, BLOCK_SIZE, 1);

    hipEventRecord(tic);
    // aes_encrypt_naive<<<B, T>>> (c_image, bitmap_info.biSizeImage, key); // encrypt the image
    aes_encrypt_shared<<<B, T>>>(c_image, bitmap_info.biSizeImage, key);
    hipEventRecord(toc);

    printf("\n");
    hipEventSynchronize(toc);
    hipEventElapsedTime(&encode_elapsed_time, tic, toc);
    encode_elapsed_time /= 1.e3;
    printf("Encryption Time: %3.6f sec \n", encode_elapsed_time);
    printf("Encryption Throughput: %3.2f MB/s\n", (double)(bitmap_info.biSizeImage/1.e6)/(encode_elapsed_time));
    printf("\n");

    // Copy encrypted image from device to host
    hipMemcpy(h_image, c_image, bitmap_info.biSizeImage, hipMemcpyDeviceToHost);
    ReloadBitmapFile(encrypted_path, h_image, &bitmap_file, &bitmap_info);

    // load encrypted image
    h_image = LoadBitmapFile(encrypted_path, &bitmap_info, &bitmap_file);

    // Copy encrypted image from host to device
    hipMemcpy(c_image, h_image, bitmap_info.biSizeImage, hipMemcpyHostToDevice);

    hipEventRecord(tic);
    // aes_decrypt_naive<<<B, T>>> (c_image, bitmap_info.biSizeImage, key); // decrypt the image
    aes_decrypt_shared<<<B, T>>>(c_image, bitmap_info.biSizeImage, key);
    hipEventRecord(toc);

    hipEventSynchronize(toc);
    hipEventElapsedTime(&decode_elapsed_time, tic, toc);
    decode_elapsed_time /= 1000;
    printf("Decryption Time: %3.6f sec\n", decode_elapsed_time);
    printf("Decryption Throughput: %3.2f MB/s", (double)(bitmap_info.biSizeImage/1.e6)/(decode_elapsed_time));
    printf("\n");

    // Copy decrypted image from device to host
    hipMemcpy(h_image, c_image, bitmap_info.biSizeImage, hipMemcpyDeviceToHost);
    ReloadBitmapFile(decrypted_path, h_image, &bitmap_file, &bitmap_info);

    hipFree(c_image);
    free(h_image);

    printf("\n");

    return 0;
}
