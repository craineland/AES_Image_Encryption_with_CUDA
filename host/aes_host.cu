#include "hip/hip_runtime.h"
// System Imports
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

// Project Imports
#include "tools/sbox.c"
#include "tools/helper_functions.c"
#include "tools/round_keys_serial.c"
#include "tools/bitmap.c"

/*
 * Serial implementation of Advanced Encryption Standard (AES)
 * Based on "aes.c" by Dani Huertas
 *
 * Encryption/Decryption process simplified with integrated
 * byte substitution and additive key rounds
 *
 * @author      Camden Landis (craine)
 * @version     05.12.2021
 *              DD.MM.YYYY
 */
// =========================================================================


// Number of columns (32-bit words) comprising the State. 
// For this standard, num. of col. = 4.
static int col_num = 4;


/*
 * Transformation in the Cipher that processes the State by cyclically 
 * shifting the last three rows of the State by different offsets. 
 */
void shift_rows(unsigned char *state) {
    unsigned char i, j, k, state_update;
    for (i = 1; i < 4; i++) {
        k = 0;
        while (k < i) {
            state_update = state[col_num*i + 0];
            
            for (j = 1; j < col_num; j++) {
                state[col_num*i + j-1] = state[col_num*i + j];
            }

            state[col_num*i + col_num-1] = state_update;
            k++;
        }
    }
}


/*
 * Transformation in the Inverse Cipher that is the  
 * cryptographic inverse of shift_rows().
 */
void inverse_shift_rows(unsigned char *state) {
    unsigned char i, j, k, state_update;
    for (i = 1; i < 4; i++) {
        k = 0;
        while (k < i) {
            state_update = state[col_num*i + col_num-1];
            
            for (j = col_num-1; j > 0; j--) {
                state[col_num*i + j] = state[col_num*i + j-1];
            }

            state[col_num*i + 0] = state_update;
            k++;
        }
    }
}


/*
 * Transformation in the Cipher that takes all of the columns of the 
 * State and mixes their data (independently of one another) to 
 * produce new columns.
 */
void mix_columns(unsigned char *state) {

    unsigned char a[] = {0x02, 0x01, 0x01, 0x03}; // a(x) = {02} + {01}x + {01}x2 + {03}x3
    unsigned char i, j, col[4], result[4];

    for (j = 0; j < col_num; j++) {
        for (i = 0; i < 4; i++) {
            col[i] = state[col_num*i+j];
        }

        coef_mult(a, col, result);

        for (i = 0; i < 4; i++) {
            state[col_num*i+j] = result[i];
        }
    }
}


/*
 * Transformation in the Inverse Cipher that is the 
 * cryptographic inverse of mix_columns().
 */
void inverse_mix_columns(unsigned char *state) {

    unsigned char a[] = {0x0e, 0x09, 0x0d, 0x0b}; // a(x) = {0e} + {09}x + {0d}x2 + {0b}x3
    unsigned char i, j, col[4], result[4];

    for (j = 0; j < col_num; j++) {
        for (i = 0; i < 4; i++) {
            col[i] = state[col_num*i+j];
        }

        coef_mult(a, col, result);

        for (i = 0; i < 4; i++) {
            state[col_num*i+j] = result[i];
        }
    }
}


/*
 * Performs the simplified AES encryption operation
 */
void aes_encrypt(unsigned char *image, int size, int key) {
    
    int i = 0;
    // byte substitution
    for (i = 0 ;i < size; i++)
        // index translation from SBOX
        image[i] = sbox[image[i]];

    // shift rows
    for (i = 0; i < size; i += 16)
        shift_rows(image + i);

    // mix columns
    for (i = 0; i < size; i += 16)
        mix_columns(image + i); 

    // Add round key
    for (i = 0; i < size; i += 16)
       key_xor(image + i);

}


/*
 * Performs the simplified AES decryption operation
 */
void aes_decrypt(unsigned char *image, int size, int key) {
    
    int i = 0;
    // add round key
    for (i = 0; i < size; i += 16)
       key_xor(image + i);

    // mix columns
    for (i = 0;i < size;i += 16)
        inverse_mix_columns(image + i);

    // shift rows
    for (i = 0;i < size;i += 16)
        inverse_shift_rows(image + i);

    // byte substitution
    for (i = 0; i < size; i++)
        // index translation from Inverse S-BOX
        image[i] = inverse_sbox[image[i]];
    
}


/*
 * Main function
 *
 * Arguments: 
 *      ./aes_serial [path to image] [integer key] [path to encrypted output] [path to decrypted output]
 *
 */
int main(int argc, char* argv[]) {

    // argument variables
    char* image_path     = argv[1];
    int   key            = atoi(argv[2]);
    char* encrypted_path = argv[3];
    char* decrypted_path = argv[4];

    if (argc < 5 || argc > 5) {
        printf("Error... Needed Arguments: ./aes_serial [path to image] [integer key] [path to encrypted output] [path to decrypted output]\n");
        return -1;
    }

    BITMAPINFOHEADER bitmap_info;
    BITMAPFILEHEADER bitmap_file;

    printf("\n");
    printf("=== HOST Encryption/Decryption Results ===\n");
    printf("\n");

    // load original image
    unsigned char *image;
    image = LoadBitmapFile(image_path, &bitmap_info, &bitmap_file);
    printf("Size of Input Image: %d%s\n", bitmap_info.biSizeImage, " bytes");
    printf("Dimensions of Image in Pixels (x,y): (%d,%d)\n", bitmap_info.biWidth, bitmap_info.biHeight);


    double encode_elapsed_time, decode_elapsed_time;
    
    double tic = clock();
    aes_encrypt(image, bitmap_info.biSizeImage, key); // encrypt the image
    double toc = clock();
    encode_elapsed_time = (toc-tic)/CLOCKS_PER_SEC;

    printf("\n");
    printf("Image Encryption Time: %3.6f sec\n", encode_elapsed_time);
    printf("Encryption Throughput: %3.2f MB/s\n", (double)(bitmap_info.biSizeImage/1.e6)/(encode_elapsed_time));
    ReloadBitmapFile(encrypted_path, image, &bitmap_file, &bitmap_info);

    // load encrypted image
    image = LoadBitmapFile(encrypted_path, &bitmap_info, &bitmap_file);

    tic = clock();
    aes_decrypt(image, bitmap_info.biSizeImage, key); // decrypt the image
    toc = clock();
    decode_elapsed_time = (toc-tic)/CLOCKS_PER_SEC;

    printf("\nImage Decryption Time: %3.6f sec\n", decode_elapsed_time);
    printf("Decryption Throughput: %3.2f MB/s\n", (double)(bitmap_info.biSizeImage/1.e6)/(decode_elapsed_time));
    ReloadBitmapFile(decrypted_path, image, &bitmap_file, &bitmap_info);

    printf("\n");

    return 0;
}
