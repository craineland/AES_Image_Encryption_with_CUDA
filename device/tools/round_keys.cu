
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// =========================================================================

//Round Keys
__device__ unsigned char key[16] = {
    0x00, 0x01, 0x02, 0x03,
    0x04, 0x05, 0x06, 0x07,
    0x08, 0x09, 0x0a, 0x0b,
    0x0c, 0x0d, 0x0e, 0x0f
};

/*
 * Key is added to the State using an XOR operation.
 */
__device__ void key_xor(unsigned char *state){

    for(int i=0;i < 16;i++) {
       state[i] = state[i]^key[i];
    }

}